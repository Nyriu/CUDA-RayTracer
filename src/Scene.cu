#include "hip/hip_runtime.h"
#include "Scene.h"
#include <iostream>
#include <ostream>
#include <hip/hip_cooperative_groups.h>

void Scene::addShape(ImplicitShape* shape) {
  shapes_num_++;
  shapes_.push_back(shape);
}


void Scene::addLight(Light* light) {
  lights_num_++;
  lights_.push_back(light);
}

void Scene::addAmbientLight(AmbientLight* light) {
  ambientLight_ = light;
}

__device__ ImplicitShape *d_shapes = nullptr;
__device__ int d_n_shapes = 0;
static __global__ void copy_kernel(ImplicitShapeInfo *infos, int n_shapes) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  if (x >= n_shapes) return;
  if (x == 0) {
    // only the first thread inits vars
    //printf("(%d) init...\n", x);
    size_t pols_size = sizeof(ImplicitShape)*n_shapes;
    d_shapes = (ImplicitShape*) malloc(pols_size);
    d_n_shapes = n_shapes;
    //printf("(%d) d_shapes=%p\n", x,d_shapes);
  }
  //__syncthreads(); // all threads must wait
  cooperative_groups::this_thread_block().sync(); // alternative // does the same

  ImplicitShape *sh_ptr = nullptr;
  ImplicitShapeInfo *isi_ptr = infos+x;

  //printf("(%d) type=%d\n", x, isi_ptr->shape_type);

  if (isi_ptr->shape_type == ShapeType::sphere) {
    //printf("(%d) we've got a sphere!\n", x);
    sh_ptr = new Sphere(*isi_ptr);
  } else if (isi_ptr->shape_type == ShapeType::none) {
    //printf("(%d) we've got a none!\n", x);
    sh_ptr = new ImplicitShape(*isi_ptr);
  } else {
    printf("(%d) we've got a PROBLEM!\n", x);
  }
  memcpy(d_shapes+x, sh_ptr, sizeof(*sh_ptr));
}

__device__ ImplicitShape* Scene::getShapes() const { return d_shapes; }




__host__ void Scene::shapes_to_device() {
  size_t infos_size = sizeof(ImplicitShapeInfo)*shapes_.size();
  ImplicitShapeInfo *infos = (ImplicitShapeInfo *) malloc(infos_size);
  int i = 0;
  for (const ImplicitShape *sh : shapes_) {
    ImplicitShapeInfo isi = sh->get_info();
    memcpy(&infos[i], &isi, sizeof(isi));
    i++;
  }

  ImplicitShapeInfo *dev_infos = nullptr;
  HANDLE_ERROR(
      hipMalloc((void**)&dev_infos, infos_size)
      );
  HANDLE_ERROR(
      hipMemcpy((void*)dev_infos, (void*)infos, infos_size, hipMemcpyHostToDevice)
      );

  free(infos);
  copy_kernel<<<1,shapes_.size()>>>(dev_infos, shapes_.size());
  HANDLE_ERROR(hipDeviceSynchronize());
  //std::cout << "here" <<  std::endl;
  //std::cout << devShapes_ <<  std::endl;
  //std::cout << d_shapes <<  std::endl;
  //std::cout << sizeof(d_shapes) <<  std::endl;
  //HANDLE_ERROR(
  //    hipMemcpy((void*)devShapes_, (void*)d_shapes, sizeof(&devShapes_), hipMemcpyDeviceToHost)
  //    );
  //std::cout << "after here" <<  std::endl;

  /// OLD STUFF BELOW
  /// PROBABLY REMOVE ALL
  /// /// DEBUG
  /// std::cout << 
  ///   "\nsizeof(SceneObject) = " << sizeof(SceneObject) << 
  ///   "\nsizeof(ImplicitShape) = " << sizeof(ImplicitShape) << 
  ///   "\nsizeof(Sphere) = " << sizeof(Sphere) << 
  ///   std::endl;
  /// /// END /// DEBUG


  /// size_t total_size = 0;
  /// for (const ImplicitShape *shape : shapes_) {
  ///   total_size += sizeof(*shape);
  ///   std::cout << 
  ///     "\nsizeof(shape) = " << sizeof(*shape) << 
  ///     std::endl;
  /// }
  /// // Static allocation on device memory
  /// HANDLE_ERROR(
  ///     hipMalloc((void**)&devShapes_, total_size)
  ///     );

  /// int offset = 0;
  /// for (const ImplicitShape *shape : shapes_) {
  ///   // Copy from host to device
  ///   HANDLE_ERROR(
  ///       hipMemcpy((void*)(devShapes_+offset), (void*)shape, sizeof(*shape), hipMemcpyHostToDevice)
  ///       );
  ///   offset++;
  /// }

  /// if (offset != shapes_num_) {
  ///   std::cout << "ERROR"
  ///     "offset = " << offset <<
  ///     "shapes_num_ = " << shapes_num_ <<
  ///     std::endl;
  ///   exit(1);
  /// }
}

__host__ void Scene::lights_to_device() {
  if (lights_num_ > 0) {
    size_t total_size = 0;
    for (Light *lgt : lights_) {
      total_size += sizeof(*lgt);
    }
    // Static allocation on device memory
    HANDLE_ERROR(
        hipMalloc((void**)&devLights_, total_size)
        );

    int offset = 0;
    for (Light *lgt : lights_) {
      // Copy from host to device
      HANDLE_ERROR(
          hipMemcpy((void*)(devLights_+offset), (void*)lgt, sizeof(*lgt), hipMemcpyHostToDevice)
          );
      offset++;
    }

    if (offset != lights_num_) {
      std::cout << "ERROR"
        "offset = " << offset <<
        "lights_num_ = " << lights_num_ <<
        std::endl;
      exit(1);
    }
  }
  if (hasAmbientLight()) {
    HANDLE_ERROR(
        hipMalloc((void**)&devAmbLight_, sizeof(AmbientLight))
        );
    HANDLE_ERROR(
        hipMemcpy((void*)devAmbLight_, (void*)ambientLight_, sizeof(AmbientLight), hipMemcpyHostToDevice)
        );
  }
}

__host__ Scene* Scene::to_device() {
  if (shapes_num_ > 0) {
    shapes_to_device();
  }
  if (lights_num_ > 0) {
    lights_to_device();
  }

  // Static allocation on device memory
  HANDLE_ERROR(
      hipMalloc((void**)&devPtr_, sizeof(Scene))
      );
  // Copy from host to device
  HANDLE_ERROR(
      hipMemcpy((void*)devPtr_, (void*)this, sizeof(Scene), hipMemcpyHostToDevice)
      );
  return devPtr_;
}
