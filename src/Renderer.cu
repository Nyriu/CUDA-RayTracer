#include "hip/hip_runtime.h"
#include "Renderer.h"
#include "common.h"
#include <chrono>
#include <iostream>


static __global__ void kernel(uchar4 *ptr,
    const Camera *cam,
    const Scene *sce,
    const Tracer *trc//, const float2 *AA_array, const int AA_array_len
    ) {

  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  // in img coord (0,0) is bottom-left
  // Put coords in [0,1]
  float u = (x + .5) / ((float) IMG_W -1); // NDC Coord
  float v = (y + .5) / ((float) IMG_H -1); // NDC Coord

  // base color
  Ray r = cam->generate_ray(u,v);
  color c = trc->trace(&r, sce);

  /// // TENTATIVO AA
  ///for (int i=0; i<AA_array_len; i++) {
  ///  Ray aar = cam->generate_ray(
  ///      (x + .5 + AA_array[i].x) / ((float) IMG_W -1),
  ///      (y + .5 + AA_array[i].y) / ((float) IMG_H -1)
  ///      );
  ///  c += trc->trace(&aar, sce);
  ///}
  ///c /= color(AA_array_len+1);
  /// END // TENTATIVO AA

  //color c(0.2);

  // accessing uchar4 vs unsigned char*
  ptr[offset].x = (int) (255 * c.r); // (int) (u * 255); //0;
  ptr[offset].y = (int) (255 * c.g); // (int) (v * 255); //(int)255/2;
  ptr[offset].z = (int) (255 * c.b); // 0;
  ptr[offset].w = 255;
}

static __global__ void kernel_update_scene(Scene *sce) {
  if (
      threadIdx.x + blockIdx.x * blockDim.x +
      threadIdx.y + blockIdx.y * blockDim.y +
      threadIdx.z + blockIdx.z * blockDim.z == 0) {
    sce->update();
  }
}

__host__ Renderer::Renderer(
    Camera *cam,
    Scene *sce,
    int max_num_tick
    ) :
  max_num_tick_(max_num_tick) {
  if (
      devCamPtr_ == nullptr ||
      devScePtr_ == nullptr //|| devTrcPtr_ == nullptr)
    ) { // a bit ugly...
    HANDLE_ERROR(
        hipMalloc((void**)&devCamPtr_, sizeof(Camera))
        );
    HANDLE_ERROR(
        hipMemcpy((void*)devCamPtr_, (void*)cam, sizeof(Camera), hipMemcpyHostToDevice)
        );
    devScePtr_ = sce->to_device();
  }
}

__host__ void Renderer::render(uchar4 *devPtr) {
  // --- Generate One Frame ---
  // TODO dims
  dim3 grids(IMG_W/16, IMG_H/16);
  dim3 threads(16,16);
  //dim3 grids(IMG_W, IMG_H);
  //dim3 threads(1);
  //float grids = 1;
  //dim3 threads(IMG_W, IMG_H);

  if (!done_cuda_free_ && current_tick_ == max_num_tick_) {
    HANDLE_ERROR(hipFree((void*)devCamPtr_));
    HANDLE_ERROR(hipFree((void*)devScePtr_));
    //HANDLE_ERROR(hipFree((void*)devTrcPtr_)); // TODO
    done_cuda_free_ = true;
    return;
  }
  if (done_cuda_free_) return;

  if (
      devCamPtr_ == nullptr ||
      devScePtr_ == nullptr //|| devTrcPtr_ == nullptr
     ) { // a bit ugly
    std::cout << "\nRenderer::render : ERROR bad device initialization?" << std::endl;
  }

  std::chrono::steady_clock::time_point t0_update = std::chrono::steady_clock::now();
  kernel_update_scene<<<1,1>>>(devScePtr_);
  HANDLE_ERROR(hipDeviceSynchronize());
  std::chrono::steady_clock::time_point t1_update = std::chrono::steady_clock::now();
  if (verbose_) {
    std::cout << "Update Time = " << std::chrono::duration_cast<std::chrono::     seconds>(t1_update - t0_update).count() << "[s]" << std::endl;
    std::cout << "Update Time = " << std::chrono::duration_cast<std::chrono::microseconds>(t1_update - t0_update).count() << "[µs]" << std::endl;
  }



  {
    /// TENTATIVO AA
    /// int AA_array_len = 8;
    /// size_t AA_array_size = sizeof(float2) * AA_array_len;
    /// float2 *AA_array = (float2 *) malloc(AA_array_size);
    /// float val = 0; //0.0000001;
    /// AA_array[0] = make_float2(-val, -val);
    /// AA_array[1] = make_float2(-val,  val);
    /// AA_array[2] = make_float2( val, -val);
    /// AA_array[3] = make_float2( val,  val);

    /// AA_array[0+4] = make_float2(AA_array[0+4].x/2.f, AA_array[0+4].y/2.f);
    /// AA_array[1+4] = make_float2(AA_array[1+4].x/2.f, AA_array[1+4].y/2.f);
    /// AA_array[2+4] = make_float2(AA_array[2+4].x/2.f, AA_array[2+4].y/2.f);
    /// AA_array[3+4] = make_float2(AA_array[3+4].x/2.f, AA_array[3+4].y/2.f);

    /// float2 *dev_AA_array = nullptr;
    /// HANDLE_ERROR( hipMalloc((void**)&dev_AA_array, AA_array_size) );
    /// HANDLE_ERROR(
    ///     hipMemcpy((void*)dev_AA_array, (void*)AA_array, AA_array_size, hipMemcpyHostToDevice)
    ///     );
    /// END // TENTATIVO AA
  }

  // qua t0
  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  //std::cout <<
  //  "generating frame num " <<
  //  current_tick_ << "\n" << std::endl;

  kernel<<<grids,threads>>>(devPtr, devCamPtr_, devScePtr_, devTrcPtr_//,
      //dev_AA_array, AA_array_len
      );
  HANDLE_ERROR(hipDeviceSynchronize());
  // qua t1
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  if (verbose_) {
    std::cout << "Frame Gen Time = " << std::chrono::duration_cast<std::chrono::     seconds>(end - begin).count() << "[s]" << std::endl;
    std::cout << "Frame Gen Time = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
  }


  //HANDLE_ERROR(hipFree((void*)devCamPtr_));
  //HANDLE_ERROR(hipFree((void*)devScePtr_));
  //HANDLE_ERROR(hipFree((void*)devTrcPtr_));

  HANDLE_ERROR(hipDeviceSynchronize());

  current_tick_++;
}

